#include "hip/hip_runtime.h"
//#include "calc_losses.cuh" 
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

unsigned sdiv (unsigned a, unsigned b) 
{
    return (a+b-1)/b;
}

//this is based off of Nicole Patterson's right up on Signal Propagation Equations for ITM
//this function (insert here) takes arrays a parameters, each index representing a single examples
//and returns a array of power losses, each eantry being the result for a single example

//this functions calculates a vaiable h
//as far as I can tell this is alse C_obs
//h_0 is the height of the obstruction in METERS
//h_1 is the height of the transmitter in METERS
//h_2 is the height of the receiver in METERS
//d_1 is the distance from the transmitter the obstruction point in KILOMETERS
//d_2 is the distance from the obstruction to the receiver in KILOMETERS
__device__ float calc_h(const float h_0, const float h_1, const float h_2, const float d_1, const float d_2)
{
	//h_ER is the height of the surface curvature at the obstruction point in meters
	float h_ER = (d_1 * d_2) / 16.944;
	float h = h_0 + h_ER - h_1 - ((h_2 - h_1) / (d_1 + d_2))*d_1;
	return h;
}

//lambda is the wavelenght
//freq is the frequency in Hertz (1/s)
__device__ float calc_lambda(float freq)
{
	return (299792458.0 / freq);
}


//v is the geometry factor
//h_0 is the height of the obstruction in METERS
//h_1 is the height of the transmitter in METERS
//h_2 is the height of the receiver in METERS
//d_1 is the distance from the transmitter the obstruction point in KILOMETERS
//d_2 is the distance from the obstruction to the receiver in KILOMETERS
//freq is the frequency in Hertz (1/s)
__device__ float calc_v(const float h_0, const float h_1, const float h_2, const float d_1, const float d_2, const float freq)
{
	//lambda is the wavelenght
	float lam = calc_lambda(freq);
	float h = calc_h(h_0, h_1, h_2, d_1, d_2);
	
	//v is the geometry factor
	//using sqrtf to ensure float version.  even though nvcc will perform it's own insertion
	float v = h * sqrtf((2.0*(d_1 + d_2)) / (lam * d_1 * d_2));
	return v;
}

//R_FR is 60% of the first Fresnel Zone radius
//d_1 is the distance from the transmitter the obstruction point in KILOMETERS
//d_2 is the distance from the obstruction to the receiver in KILOMETERS	
//freq is the frequency in Hertz (1/s)
__device__ float calc_R_FR(const float d_1, const float d_2, const float freq)
{
	float f_MHz = freq / 1000000.0;
	//using sqrtf to ensure float version.  even though nvcc will perform it's own insertion
	float R_FR = 0.6*(547.533*sqrtf((d_1*d_2) / (f_MHz*(d_1 + d_2) ) ) );
	return R_FR;
}

//this is the loss for a single example
//h_0 is the height of the obstruction in METERS
//h_1 is the height of the transmitter in METERS
//h_2 is the height of the receiver in METERS
//d_1 is the distance from the transmitter the obstruction point in KILOMETERS
//d_2 is the distance from the obstruction to the receiver in KILOMETERS
//freq is the frequency in Hertz (1/s)
__device__ float calc_loss(const float h_0, const float h_1, const float h_2, const float d_1, const float d_2, const float freq)
{
	float loss = 0.0;
	
	//v is the geometry factor
	float v = calc_v(h_0, h_1, h_2, d_1, d_2, freq);
	
	//accumulate loss from these various factors
	//FSPL loss occurs in the Fresnel Zone
	if(v <= -1.0)
	{
		//assumig the base is 10
		//using the float version instead of the default double version
		//hopefully nvcc makes the appropriate replacements
		float f_GHz = freq / 1000000000.0;
		loss += 20.0 * log10f(d_1 + d_2) + 20.0 * log10f(f_GHz) + 92.45;
	}
	//LOS loss occurs when the Freznel Zone is obstructed but the LOS line remains unobstructed
	if(v > 0.0 && v < 1.0)
	{
		//C_obs is the distance betweent he LOS and the obstruction
		float C_obs = calc_h(h_0, h_1, h_2, d_1, d_2);
		//R_FR is 60% of the first Fresnel Zone radius
		float R_FR = calc_R_FR(d_1, d_2, freq);
		loss += 6.0*(1.0 - (C_obs / R_FR));
	}
	//NLOS occurs whe the LOS is obstructed
	if(v >= 0.0)
	{
		//using log10f base 10
		//using float version of both log and sqrt
		loss += 6.9 + 20.0*log10f(sqrtf((v-0.1)*(v-0.1) + 1.0) + v - 0.1);
	}
	return loss;	
}

//This would be the function to interface with
//handles the cuda kernel
//assumes these arrays are on the cpu and creates copies on the gpu
//ph_0 is the array of heights of the obstruction in METERS
//ph_1 is the array of heights of the transmitter in METERS
//ph_2 is the array of heights of the receiver in METERS
//pd_1 is the array of distances from the transmitter the obstruction point in KILOMETERS
//pd_2 is the array of distances  from the obstruction to the receiver in KILOMETERS
//pfreq is the array of frequencies in Hertz (1/s)
//pLoss is the array used for returning loss values.  this function will try to write to it but will not allocate to it
//caller should handle memory for each of these arrays
//the index for each of these entries is the specific example
__global__ void calc_losses_kernel(const float* ph_0, const float* ph_1, const float* ph_2, 
			const float* pd_1, const float* pd_2, const float* pfreq, 
			const unsigned numSamples, const unsigned numThreads, float* pLoss)
{
	unsigned gindx = threadIdx.x + blockDim.x * blockIdx.x;	
	for(unsigned i = gindx; i < numSamples; i += numThreads)
	{
		pLoss[i] = calc_loss(ph_0[i], ph_1[i], ph_2[i], pd_1[i], pd_2[i], pfreq[i]);
	}
} 

__global__ void calc_losses_dummy_kernel(const float* ph_0, const float* ph_1, const float* ph_2, 
			const float* pd_1, const float* pd_2, const float* pfreq, 
			const unsigned numSamples, const unsigned numThreads, float* pLoss)
{
	unsigned gindx = threadIdx.x + blockDim.x * blockIdx.x;	
	for(unsigned i = gindx; i < numSamples; i += numThreads)
	{
		pLoss[i] = ph_0[i] * ph_1[i] + ph_2[i] * pd_1[i] + pd_2[i] + pfreq[i];
	}
} 

//This would be the function to interface with
//handles the cuda kernel
//assumes these arrays are on the cpe and creates copies on the gpu
//ph_0 is the array of heights of the obstruction in METERS
//ph_1 is the array of heights of the transmitter in METERS
//ph_2 is the array of heights of the receiver in METERS
//pd_1 is the array of distances from the transmitter the obstruction point in KILOMETERS
//pd_2 is the array of distances  from the obstruction to the receiver in KILOMETERS
//pfreq is the array of frequencies in Hertz (1/s)
//pLoss is the array used for returning loss values
//the index for each of these entries is the specific example
extern "C" {
void calc_losses(const float *ph_0, const float *ph_1, const float *ph_2, 
				const float *pd_1, const float *pd_2, const float *pfreq, 
				const unsigned numSamples, float *pLoss)
{
	
	int deviceID;							//device id and properties
	hipGetDevice(&deviceID);
	
	hipDeviceProp_t props;						//get properties to make best use of device
	hipGetDeviceProperties(&props, deviceID);

	unsigned ThreadsPerBlock = props.warpSize * 4;		//threads per block should be some multiple warpsize or just set it to maxThreadsPerBlock
	unsigned BlocksPerGrid = props.multiProcessorCount * 2;	//blocks per grid should be some multiple of the number of streaming multiprocessors
	unsigned numThreads = BlocksPerGrid * ThreadsPerBlock;
	
	float *ph_0_dev, *ph_1_dev, *ph_2_dev, *pd_1_dev, *pd_2_dev, *pfreq_dev, *pLoss_dev;
	size_t mem_size = numSamples * sizeof(float);
	
	hipError_t status;
	// -- gpu alloc --
	status = hipMalloc((void**)&ph_0_dev, mem_size);
	status = hipMalloc((void**)&ph_1_dev, mem_size);
	status = hipMalloc((void**)&ph_2_dev, mem_size);
	status = hipMalloc((void**)&pd_1_dev, mem_size);
	status = hipMalloc((void**)&pd_2_dev, mem_size);
	status = hipMalloc((void**)&pfreq_dev, mem_size);
	status = hipMalloc((void**)&pLoss_dev, mem_size);	//the results
	
	// -- CREATE STREAMS --
	const unsigned n_streams = 32;
	hipStream_t streams[n_streams];
	for(unsigned stream = 0; stream < n_streams; stream++)
	{
		hipStreamCreate(&streams[stream]);
	}
	
	const unsigned chunk_size = sdiv(numSamples, n_streams);
	
	for (unsigned stream = 0; stream < n_streams; stream++)
	{
		const unsigned lower = chunk_size * stream;
		const unsigned upper = min(lower + chunk_size, numSamples);
		const unsigned width = upper - lower;
		size_t mem_size_actual = sizeof(float) * width; //since they are all floats
		
		// -- copy input to gpu--
		hipMemcpyAsync(ph_0_dev + lower, ph_0 + lower, mem_size_actual, hipMemcpyHostToDevice, streams[stream]);
		hipMemcpyAsync(ph_1_dev + lower, ph_1 + lower, mem_size_actual, hipMemcpyHostToDevice, streams[stream]);
		hipMemcpyAsync(ph_2_dev + lower, ph_2 + lower, mem_size_actual, hipMemcpyHostToDevice, streams[stream]);
		hipMemcpyAsync(pd_1_dev + lower, pd_1 + lower, mem_size_actual, hipMemcpyHostToDevice, streams[stream]);
		hipMemcpyAsync(pd_2_dev + lower, pd_2 + lower, mem_size_actual, hipMemcpyHostToDevice, streams[stream]);
		hipMemcpyAsync(pfreq_dev + lower, pfreq + lower, mem_size_actual, hipMemcpyHostToDevice, streams[stream]);
		
		// -- launch per stream -- 
		//replace num_samples with width etc
		calc_losses_kernel<<<BlocksPerGrid, ThreadsPerBlock, 0, streams[stream]>>>
			(ph_0_dev + lower, ph_1_dev + lower, ph_2_dev + lower, pd_1_dev + lower, pd_2_dev + lower, pfreq_dev + lower, width, numThreads, pLoss_dev + lower);
			
		// -- copy results to cpu --
		hipMemcpyAsync(pLoss + lower, pLoss_dev + lower, mem_size_actual, hipMemcpyDeviceToHost, streams[stream]);
	}
	status = hipGetLastError();
	
	//need to let work end before moving on
	for(unsigned stream = 0; stream < n_streams; stream++)
	{
		hipStreamSynchronize(streams[stream]);
	}
	
	for(unsigned stream = 0; stream < n_streams; stream++)
	{
		hipStreamDestroy(streams[stream]);
	}
	status = hipGetLastError();
	
	//don't need to the cuda memory anymore
	hipFree(pLoss_dev);
	hipFree(pfreq_dev);
	hipFree(pd_2_dev);
	hipFree(pd_1_dev);
	hipFree(ph_2_dev);
	hipFree(ph_1_dev);
	hipFree(ph_0_dev);
} 
}

